#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "exponentialIntegral_gpu.cuh"

/**
 * @brief A dummy CUDA kernel placeholder (not used in final implementation).
 */
__global__ void dummy_kernel() {
    // No operation
}

/**
 * @brief Device function that evaluates the exponential integral E_n(x) in float precision.
 *
 * Implements continued fraction or series expansion depending on the value of x.
 *
 * @param n The order of the exponential integral.
 * @param x The point at which to evaluate E_n(x).
 * @param maxIterations Maximum number of iterations for convergence.
 * @return The value of E_n(x) as a float.
 */
__device__ float exponentialIntegralFloatDevice(const int n, const float x, const int maxIterations) {
	const float eulerConstant = 0.5772156649015329f;
	const float epsilon = 1.E-30f;
	const float bigfloat = 3.402823466e+38F; // float max
	int i, ii, nm1 = n - 1;
	float a, b, c, d, del, fact, h, psi, ans = 0.0f;

	if (n == 0) return expf(-x) / x;

	if (x > 1.0f) {
		b = x + n;
		c = bigfloat;
		d = 1.0f / b;
		h = d;
		for (i = 1; i <= maxIterations; i++) {
			a = -i * (nm1 + i);
			b += 2.0f;
			d = 1.0f / (a * d + b);
			c = b + a / c;
			del = c * d;
			h *= del;
			if (fabsf(del - 1.0f) <= epsilon)
				return h * expf(-x);
		}
		return h * expf(-x);
	} else {
		ans = (nm1 != 0 ? 1.0f / nm1 : -logf(x) - eulerConstant);
		fact = 1.0f;
		for (i = 1; i <= maxIterations; i++) {
			fact *= -x / i;
			if (i != nm1) {
				del = -fact / (i - nm1);
			} else {
				psi = -eulerConstant;
				for (ii = 1; ii <= nm1; ii++) psi += 1.0f / ii;
				del = fact * (-logf(x) + psi);
			}
			ans += del;
			if (fabsf(del) < fabsf(ans) * epsilon) return ans;
		}
		return ans;
	}
}

/**
 * @brief Device function that evaluates the exponential integral E_n(x) in double precision.
 *
 * @param n The order of the exponential integral.
 * @param x The point at which to evaluate E_n(x).
 * @param maxIterations Maximum number of iterations for convergence.
 * @return The value of E_n(x) as a double.
 */
__device__ double exponentialIntegralDoubleDevice(const int n, const double x, const int maxIterations) {
	const double eulerConstant = 0.5772156649015329;
	const double epsilon = 1.E-30;
	const double bigdouble = 1.7976931348623157E+308; // double max
	int i, ii, nm1 = n - 1;
	double a, b, c, d, del, fact, h, psi, ans = 0.0;

	if (n == 0) return exp(-x) / x;

	if (x > 1.0) {
		b = x + n;
		c = bigdouble;
		d = 1.0 / b;
		h = d;
		for (i = 1; i <= maxIterations; i++) {
			a = -i * (nm1 + i);
			b += 2.0;
			d = 1.0 / (a * d + b);
			c = b + a / c;
			del = c * d;
			h *= del;
			if (fabs(del - 1.0) <= epsilon)
				return h * exp(-x);
		}
		return h * exp(-x);
	} else {
		ans = (nm1 != 0 ? 1.0 / nm1 : -log(x) - eulerConstant);
		fact = 1.0;
		for (i = 1; i <= maxIterations; i++) {
			fact *= -x / i;
			if (i != nm1) {
				del = -fact / (i - nm1);
			} else {
				psi = -eulerConstant;
				for (ii = 1; ii <= nm1; ii++) psi += 1.0 / ii;
				del = fact * (-log(x) + psi);
			}
			ans += del;
			if (fabs(del) < fabs(ans) * epsilon) return ans;
		}
		return ans;
	}
}


/**
 * @brief CUDA kernel that computes exponential integrals in parallel.
 *
 * Each thread computes one E_n(x) value and stores it into the result array.
 *
 * @param n Maximum order of E_n(x).
 * @param numberOfSamples Number of x samples in the interval [a, b].
 * @param a Left bound of interval.
 * @param b Right bound of interval.
 * @param maxIterations Maximum iterations for convergence.
 * @param result Output array of size n * numberOfSamples.
 */
__global__ void computeExponentialIntegralKernel(
    int n, int numberOfSamples, float a, float b, int maxIterations, float* result) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = n * numberOfSamples;
    if (idx >= total) return;

    int i = idx / numberOfSamples + 1;  // n starts from 1
    int j = idx % numberOfSamples + 1;  // sample index

    float x = a + ((b - a) / numberOfSamples) * j;
    result[idx] = exponentialIntegralFloatDevice(i, x, maxIterations);
}

/**
 * @brief CUDA kernel that computes exponential integrals in double precision.
 *
 * @param n Maximum order of E_n(x).
 * @param numberOfSamples Number of x samples in the interval [a, b].
 * @param a Left bound of interval.
 * @param b Right bound of interval.
 * @param maxIterations Maximum iterations for convergence.
 * @param result Output array of size n * numberOfSamples.
 */
__global__ void computeExponentialIntegralDoubleKernel(
    int n, int numberOfSamples, double a, double b, int maxIterations, double* result) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = n * numberOfSamples;
    if (idx >= total) return;

    int i = idx / numberOfSamples + 1;  // n starts from 1
    int j = idx % numberOfSamples + 1;  // sample index

    double x = a + ((b - a) / numberOfSamples) * j;
    result[idx] = exponentialIntegralDoubleDevice(i, x, maxIterations);
}

/**
 * @brief Host function that allocates memory, launches kernel, retrieves and optionally prints results.
 *
 * Also includes timing of kernel + memory copy via CUDA events.
 *
 * @param n Maximum order of exponential integral.
 * @param numberOfSamples Number of samples between interval [a, b].
 * @param a Interval start.
 * @param b Interval end.
 * @param maxIterations Max iterations for numerical convergence.
 * @param timing Whether to time the GPU execution.
 * @param verbose Whether to print detailed results.
 */
void launch_cuda_integral(int n, int numberOfSamples, float a, float b, int maxIterations, bool timing, bool verbose) {
    int total = n * numberOfSamples;
    float* d_result;
    float* h_result = new float[total];

    hipMalloc((void**)&d_result, sizeof(float) * total);

    int threadsPerBlock = 256;
    int blocksPerGrid = (total + threadsPerBlock - 1) / threadsPerBlock;

    // CUDA timing events
    hipEvent_t start, stop;
    float milliseconds = 0.0f;
    if (timing) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
    }

    // Launch the kernel
    computeExponentialIntegralKernel<<<blocksPerGrid, threadsPerBlock>>>(
        n, numberOfSamples, a, b, maxIterations, d_result);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(h_result, d_result, sizeof(float) * total, hipMemcpyDeviceToHost);

    if (timing) {
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        std::cout << "[CUDA] Total GPU time (including kernel + memcpy): " << milliseconds << " ms" << std::endl;
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    if (verbose) {
        for (int i = 0; i < n; ++i) {
            for (int j = 0; j < numberOfSamples; ++j) {
                int idx = i * numberOfSamples + j;
                float x = a + ((b - a) / numberOfSamples) * (j + 1);
                std::cout << "[GPU] E_" << (i + 1) << "(" << x << ") = "
                          << h_result[idx] << std::endl;
            }
        }
    }

    hipFree(d_result);
    delete[] h_result;
}

void test_double_kernel(int n, int numberOfSamples, double a, double b, int maxIterations) {
	int total = n * numberOfSamples;
	double* d_result;
	double* h_result = new double[total];

	hipMalloc((void**)&d_result, sizeof(double) * total);

	int threadsPerBlock = 256;
	int blocksPerGrid = (total + threadsPerBlock - 1) / threadsPerBlock;

	computeExponentialIntegralDoubleKernel<<<blocksPerGrid, threadsPerBlock>>>(
		n, numberOfSamples, a, b, maxIterations, d_result);
	hipDeviceSynchronize();

	hipMemcpy(h_result, d_result, sizeof(double) * total, hipMemcpyDeviceToHost);

	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < numberOfSamples; ++j) {
			int idx = i * numberOfSamples + j;
			double x = a + ((b - a) / numberOfSamples) * (j + 1);
			std::cout << "[GPU-DOUBLE] E_" << (i + 1) << "(" << x << ") = "
			          << h_result[idx] << std::endl;
		}
	}

	hipFree(d_result);
	delete[] h_result;
}
